
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cfenv>
#include <cassert>

__device__ __host__ float ru_cast(long long x) {
  #ifdef __CUDA_ARCH__
    printf("In ru_cast(%ld): %f\n", x, __ll2float_ru(x));
    return __ll2float_ru(x);
  #else
    int r = std::fesetround(FE_UPWARD);
    assert(r == 0);
    return static_cast<float>(x);
  #endif
}

// This works:
// __global__ void run_utility_op_gpu(float* res, long long input) {

// Not this:
__global__ void run_utility_op_gpu(float* res, int input) {
  *res = ru_cast(input);
}

int main() {
  int int_not_float = 100000001;
  float cpu_result = ru_cast(int_not_float);
  float* gpu_result;
  hipMallocManaged(&gpu_result, sizeof(float));
  run_utility_op_gpu<<<1, 1>>>(gpu_result, int_not_float);
  hipDeviceSynchronize();
  float expect = 100000008.0f;
  printf("Rounding up of int(%d) is expected to be %f\n", int_not_float, expect);
  printf("Rounding up of int(%d) on CPU is %f\n", int_not_float, cpu_result);
  printf("Rounding up of int(%d) on GPU is %f\n", int_not_float, *gpu_result);
  return 0;
}