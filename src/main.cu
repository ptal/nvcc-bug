#include "hip/hip_runtime.h"
#include <vector>
#include "ecuda/ecuda.hpp"

__global__
// NOTE: no problem with reference `ecuda::vector<double>&`
void reverseVector(typename ecuda::vector<double>::kernel_argument vec)
{}

int main( int argc, char* argv[] )
{
  ecuda::vector<double> deviceVector;
  reverseVector<<<1,1>>>(deviceVector);
  hipDeviceSynchronize();
  return 0;
}
