#include "hip/hip_runtime.h"
#include <vector>
#include "ecuda/ecuda.hpp"

const std::size_t N = 100000;

__global__
void reverseVector( typename ecuda::vector<double>::kernel_argument vec )
{
  const std::size_t t = blockIdx.x*blockDim.x+threadIdx.x;
  if( t < (vec.size()/2) ) {
    const std::size_t u = vec.size()-t-1;
    ecuda::swap( vec[t], vec[u] );
  }
}

int main( int argc, char* argv[] )
{
  const std::size_t THREADS = 512;
  ecuda::vector<double> deviceVector( N );
  std::vector<double> hostVector( N );
  // ... initialize host vector values
  ecuda::copy( hostVector.begin(), hostVector.end(), deviceVector.begin() );
  CUDA_CALL_KERNEL_AND_WAIT( reverseVector<<<((N+THREADS-1)/THREADS),THREADS>>>( deviceVector ) );
  ecuda::copy( deviceVector.begin(), deviceVector.end(), hostVector.begin() );
  // ... host vector now contains result
  return 0;
}
